//
// Created by Felipe on 6-12-22.
//
#include "auxf.cuh"

void CpuInterfaceV3::AllocDEMHost(float *&outputD, float *&input0, float *&input1, float *&input2, float *&input3, int dim) {
    CudaSafeCall( hipHostAlloc(&input0, dim* sizeof(*input0), hipHostMallocPortable) );
    CudaSafeCall( hipHostAlloc(&input1, dim* sizeof(*input1), hipHostMallocPortable) );
    CudaSafeCall( hipHostAlloc(&input2, dim* sizeof(*input2), hipHostMallocPortable) );
    CudaSafeCall( hipHostAlloc(&input3, dim* sizeof(*input3), hipHostMallocPortable) );
    CudaSafeCall( hipHostAlloc(&outputD, dim * sizeof(*outputD), hipHostMallocPortable) );
}

void CpuInterfaceV3::FreeHostMemory(float *&out, float *&input0, float *&input1, float *&input2, float *&input3 ) {

    CudaSafeCall( hipHostFree(out) );
    CudaSafeCall( hipHostFree(input0) );
    CudaSafeCall( hipHostFree(input1) );
    CudaSafeCall( hipHostFree(input2) );
    CudaSafeCall( hipHostFree(input3) );
}

GpuInterfaceV3::GpuInterfaceV3(int dimy, int dimx) {

    this->dimy = dimy;
    this->dimx = dimx;
}

void GpuInterfaceV3::GetNumberGPUs(int &devCount) {

    hipGetDeviceCount(&devCount);
}

