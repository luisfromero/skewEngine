//
// Created by Felipe on 6-12-22.
//
#include "auxf.cuh"




void CpuInterfaceV3::AllocDEMHost(float *&outputD, float *&input0, float *&input1, float *&input2, float *&input3, int dim) {
    CudaSafeCall( hipHostAlloc(&input0, dim* sizeof(*input0), hipHostMallocPortable) );
    CudaSafeCall( hipHostAlloc(&input1, dim* sizeof(*input1), hipHostMallocPortable) );
    CudaSafeCall( hipHostAlloc(&input2, dim* sizeof(*input2), hipHostMallocPortable) );
    CudaSafeCall( hipHostAlloc(&input3, dim* sizeof(*input3), hipHostMallocPortable) );
    CudaSafeCall( hipHostAlloc(&outputD, dim * sizeof(*outputD), hipHostMallocPortable) );
}
void CpuInterfaceV3::AllocDEMHost(double *&heights, float *&h_DEM, float *&h_sDEM, float *&h_rotatedVS, float *&h_sectorVS, float **&h_multiSector, int devCount, float *&h_totalVS) {

    CudaSafeCall( hipHostAlloc(&h_DEM, dimy * dimx * sizeof(*h_DEM), hipHostMallocPortable) );
    CudaSafeCall( hipHostAlloc(&h_sDEM, 2 * dimy * dimx * sizeof(*h_sDEM), hipHostMallocPortable) );

    CudaSafeCall( hipHostAlloc(&h_rotatedVS, 2 * dimy * dimx * sizeof(*h_rotatedVS), hipHostMallocPortable) );
    CudaSafeCall( hipHostAlloc(&h_sectorVS, dimy * dimx * sizeof(*h_sectorVS), hipHostMallocPortable) );

    CudaSafeCall( hipHostAlloc(&h_multiSector, devCount * sizeof(float*), hipHostMallocPortable) );
    for (int i = 0; i < devCount; i++)
        CudaSafeCall( hipHostAlloc(&h_multiSector[i], dimy * dimx * sizeof(float), hipHostMallocPortable) );

    CudaSafeCall( hipHostAlloc(&h_totalVS, dimy * dimx * sizeof(*h_totalVS), hipHostMallocPortable) );

#ifdef DEBUG
    size = dimy * dimx * (sizeof(*h_DEM) + 2 * sizeof(*h_sDEM) + 2 * sizeof(*h_rotatedVS) + sizeof(*h_sectorVS) + sizeof(*h_totalVS));
    std::cout << "Total memory allocated in host: " << size / mb << " Mb" << std::endl;
#endif
}
void CpuInterfaceV3::FreeHostMemory(float *&h_DEM, float *&h_sDEM, float *&h_rotatedVS, float *&h_sectorVS, float **&h_multiSectorVS, int devCount, float *&h_totalVS) {

    CudaSafeCall( hipHostFree(h_DEM) );
    CudaSafeCall( hipHostFree(h_sDEM) );
    CudaSafeCall( hipHostFree(h_rotatedVS) );
    CudaSafeCall( hipHostFree(h_sectorVS) );

    for (int i = 0; i < devCount; i++)
        CudaSafeCall( hipHostFree(h_multiSectorVS[i]) );
    CudaSafeCall( hipHostFree(h_multiSectorVS) );

    CudaSafeCall( hipHostFree(h_totalVS) );
}
void CpuInterfaceV3::FreeHostMemory(float *&out, float *&input0, float *&input1, float *&input2, float *&input3 ) {

    CudaSafeCall( hipHostFree(out) );
    CudaSafeCall( hipHostFree(input0) );
    CudaSafeCall( hipHostFree(input1) );
    CudaSafeCall( hipHostFree(input2) );
    CudaSafeCall( hipHostFree(input3) );
}


GpuInterfaceV3::GpuInterfaceV3(int dimy, int dimx) {

    this->dimy = dimy;
    this->dimx = dimx;
}



void GpuInterfaceV3::GetNumberGPUs(int &devCount) {

    hipGetDeviceCount(&devCount);
}

